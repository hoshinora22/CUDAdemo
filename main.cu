
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hi_gpu() {
    printf("Hello World from GPU!\n");
}

int main() {
    hi_gpu<<<1, 1>>>();
    std::cout << "Hello, World!" << std::endl;
    return 0;
}
