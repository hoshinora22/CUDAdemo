#include "hip/hip_runtime.h"
#include <iostream>
#include "src/MathUtils.h"

__global__ void hi_gpu()
{
    printf("Hello World from GPU!\n");
}


int main()
{
    // 用__global__定义的kernel是异步的，这意味着host不会等待kernel执行完就执行下一步
    hi_gpu<<<1, 1>>>();

    std::cout << "Hello, World!" << std::endl;
    return 0;
}